// RUN: dpct --format-range=none --usm-level=none -out-root %T/new-expr %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/new-expr/new-expr.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: #define NEW_STREAM new sycl::queue *
// CHECK-NEXT: #define NEW_EVENT new dpct::event_ptr
// CHECK-EMPTY:
// CHECK-NEXT: #define NEW(T) new T
#define NEW_STREAM new hipStream_t
#define NEW_EVENT new hipEvent_t

#define NEW(T) new T

void foo() {
  int n = 16;

  // CHECK: sycl::queue **stream = new sycl::queue *;
  // CHECK-NEXT: stream = new sycl::queue *();
  // CHECK-NEXT: stream = NEW_STREAM;
  // CHECK-NEXT: stream = NEW(sycl::queue *);
  // CHECK-NEXT: sycl::queue **streams = new sycl::queue *[n];
  hipStream_t *stream = new hipStream_t;
  stream = new hipStream_t();
  stream = NEW_STREAM;
  stream = NEW(hipStream_t);
  hipStream_t *streams = new hipStream_t[n];

  // CHECK: dpct::event_ptr *event = new dpct::event_ptr;
  // CHECK-NEXT: event = new dpct::event_ptr();
  // CHECK-NEXT: event = NEW_EVENT;
  // CHECK-NEXT: event = NEW(dpct::event_ptr);
  // CHECK-NEXT: dpct::event_ptr *events = new dpct::event_ptr[n];
  hipEvent_t *event = new hipEvent_t;
  event = new hipEvent_t();
  event = NEW_EVENT;
  event = NEW(hipEvent_t);
  hipEvent_t *events = new hipEvent_t[n];
}

